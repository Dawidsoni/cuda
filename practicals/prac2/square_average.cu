#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

// Task 5:
// In the case of the slower approach, the effective transfer rate was estimated to 5 Gb/s, while in the case of the
// faster approach, it was estimated to about 20 Gb/s. As Tesla graphic card was used in the conducted experiments, the
// theoretical peak capability was equal to 80 Gb/s.

__constant__ int a_value, b_value, c_value, thread_elements;

__global__ void calculate_average(float* dev_inputs, float* dev_outputs) {
    int output_index = threadIdx.x + blockIdx.x * blockDim.x;
    int first_input_index = output_index * thread_elements;
    float values_sum = 0;
    for (int i = first_input_index; i < first_input_index + thread_elements; i++) {
        values_sum += a_value * dev_inputs[i] * dev_inputs[i] + b_value * dev_inputs[i] + c_value;
    }
    dev_outputs[output_index] = values_sum / thread_elements;
}

int main(int argc, const char** argv) {
    const int host_a_value = 3.0;
    const int host_b_value = 5.0;
    const int host_c_value = 1.0;
    const int host_inputs_count = 640000;
    const int host_outputs_count = 6400;
    const int host_thread_elements = host_inputs_count / host_outputs_count;
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(a_value), &host_a_value, sizeof(host_a_value)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(b_value), &host_b_value, sizeof(host_b_value)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_value), &host_c_value, sizeof(host_c_value)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(thread_elements), &host_thread_elements, sizeof(host_thread_elements)));

    float* dev_inputs, *dev_outputs, *host_outputs;
    checkCudaErrors(hipMalloc((void**)&dev_inputs, sizeof(float) * host_inputs_count));
    checkCudaErrors(hipMalloc((void**)&dev_outputs, sizeof(float) * host_outputs_count));
    host_outputs = (float*)malloc(sizeof(float) * host_outputs_count);

    hiprandGenerator_t random_generator;
    checkCudaErrors(hiprandCreateGenerator(&random_generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(random_generator, 1234ULL));
    checkCudaErrors(hiprandGenerateNormal(random_generator, dev_inputs, host_inputs_count, 0.0f, 1.0f));

    calculate_average<<<200, 32>>>(dev_inputs, dev_outputs);
    checkCudaErrors(hipMemcpy(host_outputs, dev_outputs, sizeof(float) * host_outputs_count, hipMemcpyDeviceToHost));
    float outputs_sum = 0;
    for (int i = 0; i < host_outputs_count; i++) {
        outputs_sum += host_outputs[i];
    }
    std::cout << "\n\n/////////////////////////////////////////////////////////////\n";
    std::cout << "Mean: " << (outputs_sum / static_cast<float>(host_outputs_count)) << "\n\n";

    checkCudaErrors(hipFree(dev_inputs));
    checkCudaErrors(hipFree(dev_outputs));
    free(host_outputs);
}