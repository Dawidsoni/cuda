#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>


__constant__ int a_value, b_value, c_value, thread_elements;

// The experiments show that computing floats is 2-4 times faster than computing doubles on GPUs.

template <class T>
__global__ void calculate_average(T* inputs, T* outputs) {
    int output_index = threadIdx.x + blockIdx.x * blockDim.x;
    int first_input_index = output_index * thread_elements;
    float values_sum = 0;
    for (int i = first_input_index; i < first_input_index + thread_elements; i++) {
        float a_power = a_value, b_power = b_value, c_power = c_value;
        for (int power = 1; power <= 2; power++) {
            values_sum += a_power * inputs[i] * inputs[i] + b_power * inputs[i] + c_power;
            a_power *= a_value;
            b_power *= b_value;
            c_power *= c_value;
        }
    }
    outputs[output_index] = values_sum / thread_elements;
}


int main(int argc, const char** argv) {
    const int host_a_value = 2;
    const int host_b_value = 3;
    const int host_c_value = 4;
    const int host_inputs_count = 640000;
    const int host_outputs_count = 6400;
    const int host_thread_elements = host_inputs_count / host_outputs_count;
    hipEvent_t start, stop;
    float milli;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(a_value), &host_a_value, sizeof(host_a_value)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(b_value), &host_b_value, sizeof(host_b_value)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_value), &host_c_value, sizeof(host_c_value)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(thread_elements), &host_thread_elements, sizeof(host_thread_elements)));

    float* dev_float_inputs, *dev_float_outputs, *host_float_outputs;
    checkCudaErrors(hipMalloc((void**)&dev_float_inputs, sizeof(float) * host_inputs_count));
    checkCudaErrors(hipMalloc((void**)&dev_float_outputs, sizeof(float) * host_outputs_count));
    host_float_outputs = (float*)malloc(sizeof(float) * host_outputs_count);

    double* dev_double_inputs, *dev_double_outputs, *host_double_outputs;
    checkCudaErrors(hipMalloc((void**)&dev_double_inputs, sizeof(double) * host_inputs_count));
    checkCudaErrors(hipMalloc((void**)&dev_double_outputs, sizeof(double) * host_outputs_count));
    host_double_outputs = (double*)malloc(sizeof(double) * host_outputs_count);

    hiprandGenerator_t random_generator;
    checkCudaErrors(hiprandCreateGenerator(&random_generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(random_generator, 1234ULL));
    checkCudaErrors(hiprandGenerateNormal(random_generator, dev_float_inputs, host_inputs_count, 0.0f, 1.0f));
    checkCudaErrors(hiprandGenerateNormalDouble(random_generator, dev_double_inputs, host_inputs_count, 0.0L, 1.0L));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    calculate_average<<<200, 32>>>(dev_float_inputs, dev_float_outputs);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);
    printf("Computation time for floats: %.3f (ms)\n", milli);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    calculate_average<<<200, 32>>>(dev_double_inputs, dev_double_outputs);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);
    printf("Computation time for doubles: %.3f (ms)\n", milli);
}